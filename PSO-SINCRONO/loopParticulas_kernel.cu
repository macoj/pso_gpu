#include "hip/hip_runtime.h"
/*
 * loop particulas
 */

#ifndef LOOPPARTICULAS
#define LOOPPARTICULAS

#include "Pso.h"
//#include <math.h>
#include <hipfft/hipfft.h>
#include <cutil.h>

__device__ float rosenbrock(float*, int, int);
__device__ float sphere (float*, int, int);
__device__ float rastrigin (float*, int, int);
__device__ float griewank (float*, int, int);
__device__ float schwefel1_2 (float*, int, int);
__device__ float p16 (float*, int, int);


// loop particulas deve ser executado com N threads, onde N é o número de agentes
__global__ void
loopParticulas(float* xx, float* atual, int dimensoes, int agentes, int funcao)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int solucaobase = tx + ty*blockDim.x;//bx*gridDim.x*blockDim.x*blockDim.y + by*blockDim.x*blockDim.y + ty*blockDim.x + tx;

	int solucao = solucaobase*dimensoes;

	if (solucaobase < agentes) {
		float minval = 0;
	//	minval = rosenbrock(xx, dimensoes, solucao);

		switch (funcao) {
			case 1 :
				minval = rosenbrock(xx, dimensoes, solucao);
				break;
			case 2 :
				minval = sphere(xx, dimensoes, solucao);
				break;
			case 3 :
				minval = rastrigin(xx, dimensoes, solucao);
				break;
			case 4 :
				minval = griewank(xx, dimensoes, solucao);
				break;
			case 5 :
				minval = schwefel1_2(xx, dimensoes, solucao);
				break;
			case 6 :
				//minval = p16(xx, dimensoes, solucao);
				break;
		}

		atual[solucaobase] = minval;
	}

}
__global__ void
calculaGbest (int* gbest, float* pbest)
{
	__shared__ float pbestdobloco[30];
	__shared__ int indice[30];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int particula = tx + ty*blockDim.x;

	if (particula < 30) {

		pbestdobloco[particula] = pbest[particula];
		indice[particula] = particula;

		__syncthreads();

		for (int s = 1; s <= 16; s = s*2) {
			if (particula % 2*s == 0)
				if (particula + s < 30)
						if (pbestdobloco[indice[particula]] > pbestdobloco[indice[particula+s]])
							indice[particula] = indice[particula + s];
			__syncthreads();
		}

		/*
		0  1  2  3  4  5  6  7  8  9  10  11  12  13  14  15
		1     |     |     |     |      |       |       |
		2           |           |              |
		4                       |
		8
		*/
		if (particula == 0) *gbest = indice[0];
	}

}

__global__ void
atualizaPbestx (float* atual, float* pbest, float* pbestx, float* xx, int iteracao)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int index = bx*gridDim.x*blockDim.x*blockDim.y + by*blockDim.x*blockDim.y + ty*blockDim.x + tx;
	int agente = index / 30;
	__shared__ float pbestdobloco[30];
	__shared__ float atualdobloco[30];

	atualdobloco[agente] = atual[agente];
	pbestdobloco[agente] = pbest[agente];

	if (index < 900) {
		if (iteracao == 0) {
			pbestx[index] = xx[index];
			pbest[agente] = atual[agente];
		} else {
			if (atualdobloco[agente] < pbestdobloco[agente]) {
				pbestx[index] = xx[index];
			}

		}
	}
}

__global__ void
atualizaPbest (float* atual, float* pbest) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int agente = tx + ty*blockDim.x;
	if (agente < 30)
		if (atual[agente] < pbest[agente])
			pbest[agente] = atual[agente];
}


// OK
//UNROLLED
__device__ float rosenbrock(float* xx, int dimensao, int solucao) {
	int i;
	float result;

	result = 0.0;

	for (i = 0; i < dimensao - 1; i++)
		result += 100.0 * (xx[solucao + i + 1] - xx[solucao + i]*xx[solucao + i])
						* (xx[solucao + i + 1] - xx[solucao + i]*xx[solucao + i])
						+ (xx[solucao + i] - 1)*(xx[solucao + i] - 1);
	if (result < 0)
		result = -result;
	return result;
}



__device__ float rastrigin (float* xx, int dimensao, int solucao) {
	int i;
	float result;
	result = 0.0;
	for (i = 0; i < dimensao; i++)
		result += xx[solucao + i] * xx[solucao + i] - 10*__cosf(2*M_PI*xx[solucao + i]) + 10;
	return result;
}

__device__ float griewank (float* xx, int dimensao, int solucao) {
	int i;
	float sumaoquadrado;
	float produtorio;
	sumaoquadrado = 0.0;
	produtorio = 1.0;
	for (i = 0; i < dimensao; i++) {
		sumaoquadrado += xx[solucao + i] * xx[solucao + i];
		produtorio *= __cosf(xx[solucao + i]/sqrtf(i+1));
	}
	sumaoquadrado = sumaoquadrado/4000 - produtorio + 1;
	return sumaoquadrado;
}


// OK
__device__ float schwefel1_2 (float* xx, int dimensao, int solucao) {
	int i;

	float result, result2;
	result = 0.0;
	for (i = 0; i < dimensao; i++) {
		result2 = 0.0;
		for (int j = 0; j < i; j++)
			result2 += xx[solucao + j];
		result += result2*result2;
	}
	return result;
}


__device__ float sphere (float* xx, int dimensao, int solucao) {
	int i;
	float result;
	result = 0.0;
	for (i = 0; i < dimensao; i++)
		result += xx[solucao + i] * xx[solucao + i];
	return result;
}
#endif
