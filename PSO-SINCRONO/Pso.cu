/*
 *
 * PSO-CUDA - it's like bolt, usain.
 *
 */

// includes, system
#include <stdio.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels

#include <inicializaParticulas_kernel.cu>
#include <loopParticulas_kernel.cu>
#include <atualizaParticulas_kernel.cu>

void rodarPSO(int, int, int, int, int, float, float, float, float);
void print(float*, int, int);

int main(int argc, char** argv) {
	// $pso numero-particulas numero-dimensoes funcao-id iteracoes numero-de-vezes
	if (argc != 10) {
		printf("use: $pso numero-de-particulas numero-de-dimensoes funcao-id iteracoes numero-de-vezes IRang_L IRang_R MINX MAXX\nfuncao-id:  1: rosenbrock 2: sphere 3: rastrigin 4: griewank 5: schwefel1_2 6: p16\n");
		exit(1);
	}
	int particulas = atoi(argv[1]);
	int dimensoes = atoi(argv[2]);
	int funcao = atoi(argv[3]);
	int iteracoes = atoi(argv[4]);
	int run_no = atoi(argv[5]);
	float IRang_L = atof(argv[6]);
	float IRang_R = atof(argv[7]);
	float MINX = atof(argv[8]);
	float MAXX = atof(argv[9]);
	rodarPSO(particulas, dimensoes, funcao, iteracoes, run_no,IRang_L, IRang_R, MINX, MAXX);
}

void rodarPSO(int NUMBER_OF_AGENTS, int DIMENSION, int funcao, int MAXITER, int run_no, float IRang_L, float IRang_R, float MINX, float MAXX) {
	float MAXV = 0.5*(MAXX - MINX);
	hipSetDevice(cutGetMaxGflopsDeviceId());
	unsigned int memoriaMatrizes = (DIMENSION * NUMBER_OF_AGENTS) * sizeof(float);
	unsigned int memoriaMatriz = (NUMBER_OF_AGENTS) * sizeof(float);
	float* vx;
	float* xx;
	float* pbestx;
	float* pbest;
	float* maxx;
	float* atual;
	int* gbest;


	// alocando memoria com hipMalloc:
	cutilSafeCall(hipMalloc((void**) &gbest, sizeof(int)));
	cutilSafeCall(hipMalloc((void**) &vx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &xx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &pbestx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &pbest, memoriaMatriz));
	cutilSafeCall(hipMalloc((void**) &atual, memoriaMatriz));
	cutilSafeCall(hipMalloc((void**) &maxx, memoriaMatriz));

	// setando parametros do CUDA

	dim3 threads(16, 16);
	dim3 grid(2, 2);

	dim3 threadsLOOP(6,6);
	dim3 gridLOOP(1,1);

	for (int itera=0; itera < run_no; itera++) {
		printf("##\n");
	        // criando timer para cada run:
        	unsigned int timer = 0;
	        cutilCheckError(cutCreateTimer(&timer));
	        cutilCheckError(cutStartTimer(timer));
	        // inicializando as particulas
	        inicializaParticulas<<<grid, threads >>>(xx, vx, pbestx, gbest, DIMENSION, NUMBER_OF_AGENTS, IRang_L, IRang_R, MAXV);

	        int iter = 0;
			do {
				loopParticulas<<<gridLOOP, threadsLOOP>>>(xx, atual, DIMENSION, NUMBER_OF_AGENTS, funcao);
				atualizaPbestx<<<grid, threads>>> (atual, pbest, pbestx, xx, iter);
				atualizaPbest<<<gridLOOP, threadsLOOP>>>(atual, pbest);
				calculaGbest<<<gridLOOP, threadsLOOP>>>(gbest, pbest);
				atualizaParticulas<<<grid, threads >>>(xx, vx, pbestx, gbest, MAXV, MAXX, MINX, NUMBER_OF_AGENTS, DIMENSION);
				iter++;
			} while (iter < MAXITER);
		float* pbestHost = (float*) malloc(memoriaMatriz);
		cutilSafeCall(hipMemcpy(pbestHost, pbest, memoriaMatriz, hipMemcpyDeviceToHost));
		int* gbestHost = (int*) malloc(sizeof (int));
		cutilSafeCall(hipMemcpy(gbestHost, gbest, sizeof(int), hipMemcpyDeviceToHost));
		printf("gbest: %d : %.10f\n", *gbestHost, pbestHost[*gbestHost]);
		free(pbestHost);
		// finaliza o timer
		cutilCheckError(cutStopTimer(timer));
		printf("#%d# %f (ms) \n", MAXITER, cutGetTimerValue(timer));
		cutilCheckError(cutDeleteTimer(timer));


	}

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed");

	cutilSafeCall(hipFree(gbest));
	cutilSafeCall(hipFree(xx));
	cutilSafeCall(hipFree(vx));
	cutilSafeCall(hipFree(pbestx));
	cutilSafeCall(hipFree(pbest));
	cutilSafeCall(hipFree(atual));
	cutilSafeCall(hipFree(maxx));
	hipDeviceReset();
}

