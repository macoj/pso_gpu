#include "hip/hip_runtime.h"
/*
 * inicializa particulas
 */


#ifndef _INICIAPARTICULAS_KERNEL_H_
#define _INICIAPARTICULAS_KERNEL_H_

#include "Pso.h"
#include "NumeroRandom.cu"

__global__ void
inicializaParticulas(float* xx, float* vx, float* pbestx, int* gbest, int dimensoes, int agentes, float IRang_L, float IRang_R, float MAXV)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int index = bx*gridDim.x*blockDim.x*blockDim.y + by*blockDim.x*blockDim.y + ty*blockDim.x + tx;

	if (index < dimensoes*agentes) {
		xx[index] = (float) ((IRang_R - IRang_L) * ((float) numeroRandom(index) / (float) INT_MAX) + (float) IRang_L);
//		xx[index] = (float) (IRang_L + ((float) numeroRandom(index) / (float) INT_MAX)*
		pbestx[index] = xx[index];

		float rnd = ((float) numeroRandom(index+1) / ((float) INT_MAX));
		vx[index] = (-MAXV + rnd*(MAXV - (-MAXV)));

//		if (rnd > 0.5)
//			vx[index] = -vx[index];

//		vx[index] = 25;

		if (index == 0) *gbest = 0;

	}
}

#endif
