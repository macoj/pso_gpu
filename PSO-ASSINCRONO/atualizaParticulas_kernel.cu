#include "hip/hip_runtime.h"
/*
 * atualiza velocidades
 *
 */


#include "Pso.h"
#include "NumeroRandom.cu"

__global__ void
atualizaParticulas(float* xx, float* vx, float* pbestx, int* gbest, float MAXV, float MAXX, float MINX, int agentes, int dimensoes) {
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int index = bx*gridDim.x*blockDim.x*blockDim.y + by*blockDim.x*blockDim.y + ty*blockDim.x + tx;
	int dimensao = index % dimensoes;
	if (index < agentes*dimensoes) {
//		float factor = 0.72894;
		float factor = 0.7298437881283576;
		float c = 2.05;
		// ATUALIZA VELOCIDADE
	/*	vx[index] = factor*(vx[index] + c * ((float) numeroRandom(index)/(float) INT_MAX)*(pbestx[index]-xx[index]) +
  		                                c * ((float) numeroRandom(index+1)/ (float) INT_MAX) * (pbestx[dimensao*agentes + gbest[0]] - xx[index]));
*/
		vx[index] = factor*(vx[index] + c * ((float) numeroRandom(index)/(float) INT_MAX)*(pbestx[index]-xx[index]) +
						c * ((float) numeroRandom(index+1)/ (float) INT_MAX) * (pbestx[dimensao + gbest[0]*dimensoes] - xx[index]));
//		vx[index] = factor*(vx[index] + c * (0.5)*(pbestx[index]-xx[index]) + c * (0.5) * (pbestx[dimensao + gbest[0]*dimensoes] - xx[index]));

		// LIMITA VELOCIDADE
		if (vx[index] > MAXV)
			vx[index] = MAXV;
		else if (vx[index] < -MAXV)
			vx[index] = -MAXV;

		// ATUALIZA POSICAO
		xx[index] = xx[index] + vx[index];
		// LIMITA POSICAO
		if (xx[index] > MAXX) {
			xx[index] = MAXX;
			vx[index] = -vx[index];
		}
		if (xx[index] < MINX) {
			xx[index] = MINX;
			vx[index] = -vx[index];
		}

	}

}
