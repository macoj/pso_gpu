/*
 *
 * PSO-CUDA - it's like bolt, usain
 *
 */

// includes, system
#include <stdio.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels

#include <inicializaParticulas_kernel.cu>
#include <loopParticulas_kernel.cu>
//#include <atualizaParticulas_kernel.cu>

void rodarPSO(int, int, int, int, int, float, float, float, float);
void print(float*, int, int);

int main(int argc, char** argv) {
	// $pso numero-particulas numero-dimensoes funcao-id iteracoes numero-de-vezes
	if (argc != 10) {
		printf("use: $pso numero-de-particulas numero-de-dimensoes funcao-id iteracoes numero-de-vezes IRang_L IRang_R MINX MAXX\nfuncao-id:  1: rosenbrock 2: sphere 3: rastrigin 4: griewank 5: schwefel1_2 6: p16\n");
		exit(1);
	}
	int particulas = atoi(argv[1]);
	int dimensoes = atoi(argv[2]);
	int funcao = atoi(argv[3]);
	int iteracoes = atoi(argv[4]);
	int run_no = atoi(argv[5]);
	float IRang_L = atof(argv[6]);
	float IRang_R = atof(argv[7]);
	float MINX = atof(argv[8]);
	float MAXX = atof(argv[9]);
	rodarPSO(particulas, dimensoes, funcao, iteracoes, run_no,IRang_L, IRang_R, MINX, MAXX);
}

void rodarPSO(int NUMBER_OF_AGENTS, int DIMENSION, int funcao, int MAXITER, int run_no, float IRang_L, float IRang_R, float MINX, float MAXX) {

	float MAXV = 0.5*(MAXX - MINX);

	hipSetDevice(cutGetMaxGflopsDeviceId());

	unsigned int memoriaMatrizes = (DIMENSION * NUMBER_OF_AGENTS) * sizeof(float);
	unsigned int memoriaMatriz = (NUMBER_OF_AGENTS) * sizeof(float);
	float* vx;
	float* xx;
	float* tx;
	float* pbestx;
	float* pbest;
	//float* maxx;
	int* gbest;

	// alocando memoria com hipMalloc:
	cutilSafeCall(hipMalloc((void**) &gbest, sizeof(int)));
	cutilSafeCall(hipMalloc((void**) &vx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &xx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &tx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &pbestx, memoriaMatrizes));
	cutilSafeCall(hipMalloc((void**) &pbest, memoriaMatriz));
	//cutilSafeCall(hipMalloc((void**) &maxx, memoriaMatriz));
	// setando parametros do CUDA

//	NAO-P16:
	dim3 threads(32, 16); // 512 threads (maximum)
	dim3 grid(1, 2); // 3 x 3


	for (int itera=0; itera < run_no; itera++) {

	        // criando timer para cada run:
        	unsigned int timer = 0;
	        cutilCheckError(cutCreateTimer(&timer));
	        cutilCheckError(cutStartTimer(timer));

		// inicializando as particulas
		inicializaParticulas<<<grid, threads >>>(xx, vx, pbestx, gbest, DIMENSION, NUMBER_OF_AGENTS, IRang_L, IRang_R, MAXV);

		int iter = 0;
		do {
			loopParticulas<<<grid, threads >>>(xx, vx, pbestx, pbest, gbest, MAXV, MAXX, MINX, DIMENSION, NUMBER_OF_AGENTS, iter, funcao);

			//debug
			//pbest
			float* pbestHost = (float*) malloc(memoriaMatriz);
			cutilSafeCall(hipMemcpy(pbestHost, pbest, memoriaMatriz, hipMemcpyDeviceToHost));

			int* gbestHost = (int*) malloc(sizeof (int));
			cutilSafeCall(hipMemcpy(gbestHost, gbest, sizeof(int), hipMemcpyDeviceToHost));
			printf("#%d# %.20f\n", iter+1, pbestHost[*gbestHost]);
			free(pbestHost);
			free(gbestHost);
			iter++;
		} while (iter < MAXITER);
	}

	// check if kernel execution generated and error
	cutilCheckMsg("Kernel execution failed");
	cutilSafeCall(hipFree(xx));
	cutilSafeCall(hipFree(vx));
	cutilSafeCall(hipFree(tx));
	cutilSafeCall(hipFree(pbestx));
	cutilSafeCall(hipFree(pbest));
	cutilSafeCall(hipFree(gbest));
	//cutilSafeCall(hipFree(maxx));
	hipDeviceReset();
}

