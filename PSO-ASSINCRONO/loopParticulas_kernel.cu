#include "hip/hip_runtime.h"
/*
 * loop particulas
 */

#include "Pso.h"
//#include <math.h>
#include <hipfft/hipfft.h>
#include <cutil.h>

__device__ float rosenbrock(float*, int, int);
__device__ float sphere (float*, int, int);
__device__ float rastrigin (float*, int, int);
__device__ float griewank (float*, int, int);
__device__ float schwefel1_2 (float*, int, int);
__device__ float p16 (float*, int, int);


__global__ void
loopParticulas(float* xx, float* vx, float* pbestx, float* pbest, int* gbest, float MAXV, float MAXX, float MINX, int dimensoes, int agentes, int iteracao, int funcao)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ float minval[30];

	int index = bx*gridDim.x*blockDim.x*blockDim.y + by*blockDim.x*blockDim.y + ty*blockDim.x + tx;

	index = index - 2*(bx + by*gridDim.x);	// 2 threads perdidas por bloco
	//index = index - 12*(bx + by*gridDim.x); 	// 12 threads perdidas por bloco
	//	index = index - 4*(bx + by*3); 		// 4 threads perdidas por bloco
	//index = index - 8*(bx + by*gridDim.x); 		// 8 threads perdidas por bloco

	int id = tx + ty*blockDim.x;

	if (id >= 510) index = agentes*dimensoes;	// máximo 510/30 = 17 particulas por bloco
	//if (id >= 240) index = agentes*dimensoes;	// máximo 240/30 = 8 particulas por bloco
	//if (id >= 60) index = agentes*dimensoes;	// máximo 60/30 = 2 particulas por bloco
	//if (id >= 120) index = agentes*dimensoes;	// máximo 120/30 = 4 particulas por bloco

	//if ((id >=180) && (bx == 1) && (by == 0)) index = agentes*dimensoes;

	if (index < agentes*dimensoes) {

		int solucao = index / dimensoes;
		int pbestsolucao = solucao;

		solucao = solucao * dimensoes;

		if (index == solucao)		// apenas uma thread da particula irá executar a função
			//minval[pbestsolucao] = rosenbrock(xx, index, dimensoes, agentes, solucao);
			minval[pbestsolucao] =  schwefel1_2 (xx, dimensoes, solucao);
	//		minval[pbestsolucao] = sphere(xx, dimensoes, solucao);
			//minval[pbestsolucao] = p16(xx, dimensoes, solucao);

		/*
			switch (funcao) {
	 			case 1 :
					minval[pbestsolucao] = rosenbrock(xx, dimensoes, solucao);
					break;
				case 2 :
					minval[pbestsolucao] = sphere(xx, dimensoes, solucao);
					break;
				case 3 :
					minval[pbestsolucao] = rastrigin(xx, dimensoes, solucao);
					break;
				case 4 :
					minval[pbestsolucao] = griewank(xx, dimensoes, solucao);
					break;
				case 5 :
					minval[pbestsolucao] = schwefel1_2(xx, dimensoes, solucao);
					break;
				case 6 :
					minval[pbestsolucao] = p16(xx, dimensoes, solucao);
					break;
			}
*/
//		__syncthreads();

		if (iteracao == 0) {
			if (index == solucao)
				pbest[pbestsolucao] = minval[pbestsolucao];
			pbestx[index] = xx[index];
		} else if (minval[pbestsolucao] <= pbest[pbestsolucao]) {
			if (index == solucao)
				pbest[pbestsolucao] = minval[pbestsolucao];
			pbestx[index] = xx[index];
			if (index == solucao)
				if (pbest[pbestsolucao] < pbest[*gbest])	// RACE CONDITION?
					*gbest = pbestsolucao;
		}

		__syncthreads();
		int dimensao = index % dimensoes;
                float factor = 0.7298437881283576;
                float c = 2.05;
                // ATUALIZA VELOCIDADE
                vx[index] = factor*(vx[index] + c * ((float) numeroRandom(index)/(float) INT_MAX)*(pbestx[index]-xx[index]) +
                                                c * ((float) numeroRandom(index+1)/ (float) INT_MAX) * (pbestx[dimensao + gbest[0]*dimensoes] - xx[index]));
                // LIMITA VELOCIDADE
                if (vx[index] > MAXV)
                        vx[index] = MAXV;
                else if (vx[index] < -MAXV)
                        vx[index] = -MAXV;

                // ATUALIZA POSICAO
               xx[index] = xx[index] + vx[index];

                // LIMITA POSICAO
                if (xx[index] > MAXX) {
                        xx[index] = MAXX;
                        vx[index] = -vx[index];
                }
                if (xx[index] < MINX) {
                        xx[index] = MINX;
                        vx[index] = -vx[index];
                }

	}
}

/****************************************************
 * FUNCOES ******************************************
 ****************************************************/

// OK
//UNROLLED
__device__ float rosenbrock(float* xx, int dimensao, int solucao) {
	int i;

	float result;

	result = 0.0;

	for (i = 0; i < dimensao - 1; i++)
		result += 100.0 * (xx[solucao + i + 1] - xx[solucao + i]*xx[solucao + i])
						* (xx[solucao + i + 1] - xx[solucao + i]*xx[solucao + i])
						+ (xx[solucao + i] - 1)*(xx[solucao + i] - 1);


	if (result < 0)
		result = -result;
	return result;
}


// OK
// UNROLLED
__device__ float rastrigin (float* xx, int dimensao, int solucao) {
	int i;
	float result;
	result = 0.0;
	for (i = 0; i < dimensao; i++)
		result += xx[solucao + i] * xx[solucao + i] - 10*__cosf(2*M_PI*xx[solucao + i]) + 10;
	return result;
}

// OK
// UNROLLED
__device__ float griewank (float* xx, int dimensao, int solucao) {
	int i;
	float sumaoquadrado;
	float produtorio;
	sumaoquadrado = 0.0;
	produtorio = 1.0;
	for (i = 0; i < dimensao; i++) {
		sumaoquadrado += xx[solucao + i] * xx[solucao + i];
		produtorio *= __cosf(xx[solucao + i]/sqrtf(i+1));
	}

	return sumaoquadrado;
}


// OK
__device__ float schwefel1_2 (float* xx, int dimensao, int solucao) {
	int i;

	float result, result2;
	result = 0.0;

	for (i = 0; i < dimensao; i++) {
		result2 = 0.0;
		for (int j = 0; j < i; j++)
			result2 += xx[solucao + j];
		result += result2*result2;
	}
	return result;
}

// OK
// UNROLLED
__device__ float sphere (float* xx, int dimensao, int solucao) {
	int i;
	float result;
	result = 0.0;

	for (i = 0; i < dimensao; i++)
		result += xx[solucao + i] * xx[solucao + i];

	return result;
}

